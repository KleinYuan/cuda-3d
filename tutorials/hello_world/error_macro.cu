
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
  	fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
	assert(result == hipSuccess);
  }

  return result;
}

int main()
{
  checkCuda(hipDeviceSynchronize());
}
