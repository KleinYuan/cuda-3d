
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{
  int indexWithinTheGrid = blockIdx.x * blockDim.x + threadIdx.x;
  int gridStride = gridDim.x * blockDim.x;
  for (int i = indexWithinTheGrid; i < N; i +=gridStride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * `N` is greater than the size of the grid (see below).
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  /*
   * The size of this grid is 256*32 = 8192.
   */

  size_t threads_per_block = 256;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}

