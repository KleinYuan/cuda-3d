
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Initialize array values on the host.
 */

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

/*
 * Double elements in parallel on the GPU.
 */

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

/*
 * Check all elements have been doubled on the host.
 */

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 100;
  int *a;

  size_t size = N * sizeof(int);

  // You do this in CPU but won't work on GPU:  a = (int *)malloc(size);
  // since this memory is only accessable by the host (CPU) not the device (GPU);
  // Instead, you shall do the following:

  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 10;
  size_t number_of_blocks = 10;

  /*
   * This launch will not work until the pointer `a` is also
   * available to the device.
   */

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  // this is how you free memory in CPU: free(a);
  // instead, you shall do:
  hipFree(a);
}

