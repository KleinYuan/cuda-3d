
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess){
      fprintf(stderr, "CUDA Runtime Error: %s \n", hipGetErrorString(result));
  }
  return result;
}

__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMaxKernel(float *data, int N, float *max_val){
  int idxWithinTheGrid = threadIdx.x + blockDim.x * blockIdx.x;
  int gridStride = gridDim.x * blockDim.x;

  for(int i = idxWithinTheGrid; i < N; i += gridStride)
  {
    float old = atomicMax(max_val, data[i]);
    printf("[findMaxKernel] Perform GPU code, comparing %f (old)  and %f (new) at %p \n", old, data[i], &data[0] + sizeof(float) * i);
  }
}


int main()
{
  // Init
  const int N = 4;
  float foo [N] = {0.1, 1.1, 2.1, 3.1};
  float *host_data = &foo[0];
  float *device_data;
  size_t size = N * sizeof(float);
  float max_val = 0.f;
  float *device_max;

  hipMallocManaged(&device_data, size);
  hipMallocManaged(&device_max, sizeof(float));
  // Copy foo -> host_data to device_data
  hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);
  hipMemcpy(device_max, &max_val, sizeof(float), hipMemcpyHostToDevice);
  // Run a custom kernel to find the max val in foo, adding the index to the element
  findMaxKernel<<<4, 1>>>(device_data, N, device_max);
  hipMemcpy(&max_val, device_max, sizeof(float), hipMemcpyDeviceToHost);
  // Sync
  checkCuda(hipDeviceSynchronize());
  printf("max_val is %lf\n", max_val);
  hipFree(device_data);
}
