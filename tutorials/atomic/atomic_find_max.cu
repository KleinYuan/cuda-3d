
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess){
      fprintf(stderr, "CUDA Runtime Error: %s \n", hipGetErrorString(result));
  }
  return result;
}

__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMaxKernel(float *data, int N, float *max_val){
  int idxWithinTheGrid = threadIdx.x + blockDim.x * blockIdx.x;
  int gridStride = gridDim.x * blockDim.x;

  for(int i = idxWithinTheGrid; i < N; i += gridStride)
  {
    float old = atomicMax(max_val, data[i]);
    printf("[findMaxKernel] Perform GPU code, comparing %f (old)  and %f (new) at %p \n", old, data[i], &data[0] + sizeof(float) * i);
  }
}

__global__ void findMaxAndOffsetKernel(float *data, int N, float *max_val, float *device_offset){
  int idxWithinTheGrid = threadIdx.x + blockDim.x * blockIdx.x;
  int gridStride = gridDim.x * blockDim.x;

  for(int i = idxWithinTheGrid; i < N; i += gridStride)
  {
    float old = atomicMax(max_val, data[i]);
    float old_offset = old + *device_offset;
    printf("[findMaxKernel] Perform GPU code, comparing %f (old)  and %f (new) at %p \n", old, data[i], &data[0] + sizeof(float) * i);
    printf("[findMaxKernel] After offset %f (old)  becomes %f (old_offset)  \n", old, old_offset);

  }
}

int main()
{
  // Init
  const int N = 4;
  float foo [N] = {0.1, 1.1, 2.1, 3.1};
  float *host_data = &foo[0];
  float *device_data;
  size_t size = N * sizeof(float);
  float max_val = 0.f;
  float *device_max;

  // The following offset is to show how to de-reference device pointer in a kernel
  // uncomment findMaxAndOffsetKernel to see the effect
  float offset_val = 2.f;
  float *device_offset;
  hipMalloc(&device_offset, sizeof(float));
  hipMemcpy(device_offset, &offset_val,  sizeof(float), hipMemcpyHostToDevice);


  hipMallocManaged(&device_data, size);
  hipMallocManaged(&device_max, sizeof(float));
  // Copy foo -> host_data to device_data
  hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);
  hipMemcpy(device_max, &max_val, sizeof(float), hipMemcpyHostToDevice);
  // Run a custom kernel to find the max val in foo, adding the index to the element
  findMaxKernel<<<4, 1>>>(device_data, N, device_max);
  // findMaxAndOffsetKernel<<<4, 1>>>(device_data, N, device_max, device_offset);
  hipMemcpy(&max_val, device_max, sizeof(float), hipMemcpyDeviceToHost);
  // Sync
  checkCuda(hipDeviceSynchronize());
  printf("max_val is %lf\n", max_val);
  hipFree(device_data);
  hipFree(device_max);
  hipFree(device_offset);
}
