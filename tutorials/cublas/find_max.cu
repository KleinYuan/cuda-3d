
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess){
      fprintf(stderr, "CUDA Runtime Error: %s \n", hipGetErrorString(result));
  }
  return result;
}

// https://docs.nvidia.com/cuda/cublas/index.html
int main()
{
  // Init
  const int N = 6;
  float foo [N] = {0.1, 1.1, 2.1, 3.1, 5.1, 4.1};
  float *host_data = &foo[0];
  float *device_data;
  size_t size = N * sizeof(float);
  int max_idx = 0;
  hipblasHandle_t cublas_handle_;
  hipblasCreate(&cublas_handle_);

  hipMallocManaged(&device_data, size);

  // Copy foo -> host_data to device_data
  hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);
  // Find out the max index
  hipblasIsamax(cublas_handle_, N, device_data, 1, &max_idx);
  // Sync
  checkCuda(hipDeviceSynchronize());
  printf("Max Id is %d\n", max_idx);
  hipFree(device_data);
  hipblasDestroy(cublas_handle_);
}
